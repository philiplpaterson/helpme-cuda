#include "hip/hip_runtime.h"
#include <cassert>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

template <typename Real>
__global__ void getall(Real* cudadata, Real* d_A, Real* d_W) {

        
    // unsigned long tid = blockIdx.x  threadIdx.x;
    for(int i = 0; i<3; ++i)
    {
        cudadata[blockIdx.x*3 + threadIdx.x] = d_A[blockIdx.x*3 + i] * d_A[threadIdx.x*3 + i] * d_W[i*3];
    }
    __syncthreads();
  }