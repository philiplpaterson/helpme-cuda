#include "hip/hip_runtime.h"


#include <cassert>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "helpme_standalone.cuh"
#include <mpi.h>


extern "C" void run_code(int numThreads, int myRank, int nx, int ny, int nz)
{
    const double tolerance = 1e-8;

    float kappa = 0.3;
    int gridX = 32;
    int gridY = 32;
    int gridZ = 32;
    int kMaxX = 9;
    int kMaxY = 9;
    int kMaxZ = 9;
    int splineOrder = 6;

    helpme::Matrix<double> coords(
        {{2.0, 2.0, 2.0}, {2.5, 2.0, 3.0}, {1.5, 2.0, 3.0}, {0.0, 0.0, 0.0}, {0.5, 0.0, 1.0}, {-0.5, 0.0, 1.0}});
    helpme::Matrix<double> charges({-0.834, 0.417, 0.417, -0.834, 0.417, 0.417});
    double scaleFactor = 332.0716;
    helpme::Matrix<double> serialVirial(6, 1);
    helpme::Matrix<double> serialForces(6, 3);

    // Generate a serial benchmark first
    double energyS;
    if (myRank == 0) {
        std::cout << "Num Threads " << numThreads << std::endl;
        auto pme = std::unique_ptr<PMEInstanceD>(new PMEInstanceD());
        pme->setup(1, kappa, splineOrder, gridX, gridY, gridZ, scaleFactor, numThreads);
        pme->setLatticeVectors(20, 20, 20, 90, 90, 90, PMEInstanceD::LatticeType::XAligned);
        energyS = pme->computeEFVRec(0, charges, coords, serialForces, serialVirial);
        std::cout << "Serial results:" << std::endl;
        std::cout << "Total rec energy " << energyS << std::endl;
        std::cout << "Total forces" << std::endl << serialForces << std::endl;
        std::cout << "Total virial" << std::endl << serialVirial << std::endl;
    }

    // Now the parallel version
    auto pmeP = std::unique_ptr<PMEInstanceD>(new PMEInstanceD());
    double parallelEnergy, nodeEnergy;
    helpme::Matrix<double> nodeForces(6, 3);
    helpme::Matrix<double> nodeVirial(6, 1);
    helpme::Matrix<double> parallelForces(6, 3);
    helpme::Matrix<double> parallelVirial(6, 1);

    nodeForces.setZero();
    nodeVirial.setZero();
    pmeP->setupParallel(1, kappa, splineOrder, gridX, gridY, gridZ, scaleFactor, 1, MPI_COMM_WORLD,
                        PMEInstanceD::NodeOrder::ZYX, nx, ny, nz);
    pmeP->setLatticeVectors(20, 20, 20, 90, 90, 90, PMEInstanceD::LatticeType::XAligned);
    nodeEnergy = pmeP->computeEFVRec(0, charges, coords, nodeForces, nodeVirial);
    MPI_Reduce(&nodeEnergy, &parallelEnergy, 1, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Reduce(nodeForces[0], parallelForces[0], 6 * 3, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Reduce(nodeVirial[0], parallelVirial[0], 6, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
    if (myRank == 0) {
        std::cout << "Parallel results (nProcs = " << nx << ", " << ny << ", " << nz << "):" << std::endl;
        std::cout << "Total rec energy " << parallelEnergy << std::endl;
        std::cout << "Total forces " << std::endl << parallelForces << std::endl;
        std::cout << "Total virial " << std::endl << parallelVirial << std::endl;

        assert((std::abs(energyS - parallelEnergy) < tolerance));
        assert((serialForces.almostEquals(parallelForces, tolerance)));
        assert((serialVirial.almostEquals(parallelVirial, tolerance)));
    }
    // Now the compressed version
    nodeForces.setZero();
    nodeVirial.setZero();
    pmeP->setupCompressedParallel(1, kappa, splineOrder, gridX, gridY, gridZ, kMaxX, kMaxY, kMaxZ, scaleFactor, 1,
                                  MPI_COMM_WORLD, PMEInstanceD::NodeOrder::ZYX, nx, ny, nz);
    pmeP->setLatticeVectors(20, 20, 20, 90, 90, 90, PMEInstanceD::LatticeType::XAligned);
    nodeEnergy = pmeP->computeEFVRec(0, charges, coords, nodeForces, nodeVirial);
    MPI_Reduce(&nodeEnergy, &parallelEnergy, 1, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Reduce(nodeForces[0], parallelForces[0], 6 * 3, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Reduce(nodeVirial[0], parallelVirial[0], 6, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
    if (myRank == 0) {
        std::cout << std::endl << "Compressed" << std::endl;
        std::cout << "Parallel results (nProcs = " << nx << ", " << ny << ", " << nz << "):" << std::endl;
        std::cout << "Total rec energy " << parallelEnergy << std::endl;
        std::cout << "Total forces " << std::endl << parallelForces << std::endl;
        std::cout << "Total virial " << std::endl << parallelVirial << std::endl;

        assert((std::abs(energyS - parallelEnergy) < tolerance));
        assert((serialForces.almostEquals(parallelForces, tolerance)));
        assert((serialVirial.almostEquals(parallelVirial, tolerance)));
    }
    pmeP.reset();  // This ensures that the PME object cleans up its MPI data BEFORE MPI_Finalize is called;

}



